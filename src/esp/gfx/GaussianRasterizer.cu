#include "GaussianRasterizer.h"

// Standard library includes
#include <cmath>
#include <cstring>
#include <cassert>
#include <iostream>

// CUDA headers
#include <hip/hip_runtime.h>

// Forward declare only what we need - avoid including Magnum headers
namespace esp {
namespace assets {
class GaussianSplattingData;

// Simple POD struct to pass Gaussian data without Magnum types
struct GaussianSplatSimple {
  float position[3];
  float normal[3];
  float f_dc[3];
  float opacity;
  float scale[3];
  float rotation[4];  // quaternion (x, y, z, w)
};
}  // namespace assets
}  // namespace esp

// Include GL types after checking they're available
#ifdef __HIPCC__
// When compiling with nvcc, use CUDA's GL interop which defines GL types
#include <cuda_gl_interop.h>
#ifndef GL_TEXTURE_2D
#define GL_TEXTURE_2D 0x0DE1
#endif
#endif

// Include rasterizer
#include "esp/gfx/gaussian_rasterizer/rasterizer.h"

// CUDA error checking macro
#define CUDA_CHECK(call)                                                      \
  do {                                                                        \
    hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                 \
      std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << " - "  \
                << hipGetErrorString(err) << std::endl;                      \
      assert(false && "CUDA error occurred");                                 \
    }                                                                         \
  } while (0)

namespace esp {
namespace gfx {

// Helper function to allocate memory via lambda
char* allocateBuffer(void*& bufferPtr, size_t size) {
  CUDA_CHECK(hipMalloc(&bufferPtr, size));
  return reinterpret_cast<char*>(bufferPtr);
}

// Implementation struct that holds CUDA resources
struct GaussianRasterizer::Impl {
  // CUDA memory for Gaussian data
  void* d_positions = nullptr;
  void* d_normals = nullptr;
  void* d_sh_dc = nullptr;
  void* d_sh_rest = nullptr;
  void* d_opacities = nullptr;
  void* d_scales = nullptr;
  void* d_rotations = nullptr;

  // CUDA-OpenGL interop resources
  hipGraphicsResource* colorTexResource = nullptr;
  hipGraphicsResource* depthTexResource = nullptr;

  // Persistent CUDA buffers for rasterization
  void* geomBuffer = nullptr;
  void* binningBuffer = nullptr;
  void* imageBuffer = nullptr;

  int lastGaussianCount = 0;
  int lastWidth = 0;
  int lastHeight = 0;

  ~Impl() {
    cleanup();
  }

  void cleanup() {
    // Unregister OpenGL resources
    if (colorTexResource) {
      hipGraphicsUnregisterResource(colorTexResource);
      colorTexResource = nullptr;
    }
    if (depthTexResource) {
      hipGraphicsUnregisterResource(depthTexResource);
      depthTexResource = nullptr;
    }

    // Free CUDA memory
    if (d_positions) hipFree(d_positions);
    if (d_normals) hipFree(d_normals);
    if (d_sh_dc) hipFree(d_sh_dc);
    if (d_sh_rest) hipFree(d_sh_rest);
    if (d_opacities) hipFree(d_opacities);
    if (d_scales) hipFree(d_scales);
    if (d_rotations) hipFree(d_rotations);
    if (geomBuffer) hipFree(geomBuffer);
    if (binningBuffer) hipFree(binningBuffer);
    if (imageBuffer) hipFree(imageBuffer);

    d_positions = d_normals = d_sh_dc = d_sh_rest = nullptr;
    d_opacities = d_scales = d_rotations = nullptr;
    geomBuffer = binningBuffer = imageBuffer = nullptr;
  }

  void uploadGaussianData(const GaussianSplatSimple* gaussians, int P) {
    // Only reallocate if size changed
    if (P != lastGaussianCount) {
      if (d_positions) hipFree(d_positions);
      if (d_normals) hipFree(d_normals);
      if (d_sh_dc) hipFree(d_sh_dc);
      if (d_opacities) hipFree(d_opacities);
      if (d_scales) hipFree(d_scales);
      if (d_rotations) hipFree(d_rotations);

      CUDA_CHECK(hipMalloc(&d_positions, P * 3 * sizeof(float)));
      CUDA_CHECK(hipMalloc(&d_normals, P * 3 * sizeof(float)));
      CUDA_CHECK(hipMalloc(&d_sh_dc, P * 3 * sizeof(float)));
      CUDA_CHECK(hipMalloc(&d_opacities, P * sizeof(float)));
      CUDA_CHECK(hipMalloc(&d_scales, P * 3 * sizeof(float)));
      CUDA_CHECK(hipMalloc(&d_rotations, P * 4 * sizeof(float)));

      lastGaussianCount = P;
    }

    // Prepare CPU buffers
    std::vector<float> positions(P * 3);
    std::vector<float> normals(P * 3);
    std::vector<float> sh_dc(P * 3);
    std::vector<float> opacities(P);
    std::vector<float> scales(P * 3);
    std::vector<float> rotations(P * 4);

    for (int i = 0; i < P; ++i) {
      const auto& g = gaussians[i];
      positions[i * 3 + 0] = g.position[0];
      positions[i * 3 + 1] = g.position[1];
      positions[i * 3 + 2] = g.position[2];

      normals[i * 3 + 0] = g.normal[0];
      normals[i * 3 + 1] = g.normal[1];
      normals[i * 3 + 2] = g.normal[2];

      sh_dc[i * 3 + 0] = g.f_dc[0];
      sh_dc[i * 3 + 1] = g.f_dc[1];
      sh_dc[i * 3 + 2] = g.f_dc[2];

      opacities[i] = g.opacity;

      scales[i * 3 + 0] = g.scale[0];
      scales[i * 3 + 1] = g.scale[1];
      scales[i * 3 + 2] = g.scale[2];

      rotations[i * 4 + 0] = g.rotation[0];
      rotations[i * 4 + 1] = g.rotation[1];
      rotations[i * 4 + 2] = g.rotation[2];
      rotations[i * 4 + 3] = g.rotation[3];
    }

    // Upload to GPU
    CUDA_CHECK(hipMemcpy(d_positions, positions.data(), P * 3 * sizeof(float),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_normals, normals.data(), P * 3 * sizeof(float),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_sh_dc, sh_dc.data(), P * 3 * sizeof(float),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_opacities, opacities.data(), P * sizeof(float),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_scales, scales.data(), P * 3 * sizeof(float),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_rotations, rotations.data(), P * 4 * sizeof(float),
                          hipMemcpyHostToDevice));
  }
};

GaussianRasterizer::GaussianRasterizer() : impl_(std::make_unique<Impl>()) {}

GaussianRasterizer::~GaussianRasterizer() = default;

void GaussianRasterizer::render(
    const GaussianSplatSimple* gaussians,
    int numGaussians,
    const float* viewMatrix,
    const float* projMatrix,
    int width,
    int height,
    unsigned int colorTextureId,
    unsigned int depthTextureId,
    float backgroundR,
    float backgroundG,
    float backgroundB) {
  const int W = width;
  const int H = height;
  const int P = numGaussians;

  if (P == 0) {
    std::cerr << "Warning: No Gaussians to render" << std::endl;
    return;
  }

  // Upload Gaussian data to CUDA
  impl_->uploadGaussianData(gaussians, P);

  // Register OpenGL textures with CUDA (if not already registered)
  if (!impl_->colorTexResource) {
    CUDA_CHECK(hipGraphicsGLRegisterImage(
        &impl_->colorTexResource, colorTextureId,
        GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
  }
  if (!impl_->depthTexResource) {
    CUDA_CHECK(hipGraphicsGLRegisterImage(
        &impl_->depthTexResource, depthTextureId,
        GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
  }

  // Map OpenGL textures to CUDA
  CUDA_CHECK(hipGraphicsMapResources(1, &impl_->colorTexResource, 0));
  CUDA_CHECK(hipGraphicsMapResources(1, &impl_->depthTexResource, 0));

  hipArray* colorArray = nullptr;
  hipArray* depthArray = nullptr;
  CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(
      &colorArray, impl_->colorTexResource, 0, 0));
  CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(
      &depthArray, impl_->depthTexResource, 0, 0));

  // Allocate output buffers in device memory
  float* d_colorOutput = nullptr;
  float* d_depthOutput = nullptr;
  CUDA_CHECK(hipMalloc(&d_colorOutput, W * H * 3 * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_depthOutput, W * H * sizeof(float)));

  // Matrices are already in float* format (row-major)
  const float* viewmat = viewMatrix;
  const float* projmat = projMatrix;

  // Compute camera parameters
  float focal_x = projmat[0] * W / 2.0f;
  float focal_y = projmat[5] * H / 2.0f;
  float tan_fovx = 1.0f / projmat[0];
  float tan_fovy = 1.0f / projmat[5];

  // Camera position - extract from view matrix
  // For simplicity, assuming camera is at origin (can be improved)
  float cam_pos[3] = {-viewmat[12], -viewmat[13], -viewmat[14]};

  // Background color
  float bg_color[3] = {backgroundR, backgroundG, backgroundB};

  // Allocate persistent buffers using lambdas
  auto geometryBufferFunc = [this](size_t size) {
    return allocateBuffer(impl_->geomBuffer, size);
  };
  auto binningBufferFunc = [this](size_t size) {
    return allocateBuffer(impl_->binningBuffer, size);
  };
  auto imageBufferFunc = [this](size_t size) {
    return allocateBuffer(impl_->imageBuffer, size);
  };

  // Call CUDA rasterizer
  int D = 3;  // SH degree 0 (only DC component)
  int M = 0;  // No higher order SH

  CudaRasterizer::Rasterizer::forward(
      geometryBufferFunc, binningBufferFunc, imageBufferFunc,
      P, D, M, bg_color, W, H,
      reinterpret_cast<float*>(impl_->d_positions),    // means3D
      nullptr,                                         // shs (use precomp colors)
      reinterpret_cast<float*>(impl_->d_sh_dc),        // colors_precomp
      reinterpret_cast<float*>(impl_->d_opacities),    // opacities
      reinterpret_cast<float*>(impl_->d_scales),       // scales
      1.0f,                                            // scale_modifier
      reinterpret_cast<float*>(impl_->d_rotations),    // rotations
      nullptr,                                         // cov3D_precomp
      viewmat, projmat, cam_pos, tan_fovx, tan_fovy,
      false,                                           // prefiltered
      d_colorOutput,                                   // out_color
      d_depthOutput,                                   // depth (now supported!)
      false,                                           // antialiasing
      nullptr,                                         // radii
      false);                                          // debug

  // Copy color output to OpenGL texture via CUDA array
  CUDA_CHECK(hipMemcpy2DToArray(
      colorArray, 0, 0, d_colorOutput, W * 3 * sizeof(float),
      W * 3 * sizeof(float), H, hipMemcpyDeviceToDevice));

  // Copy depth output to OpenGL texture via CUDA array
  CUDA_CHECK(hipMemcpy2DToArray(
      depthArray, 0, 0, d_depthOutput, W * sizeof(float),
      W * sizeof(float), H, hipMemcpyDeviceToDevice));

  // Cleanup
  hipFree(d_colorOutput);
  hipFree(d_depthOutput);

  // Unmap resources
  CUDA_CHECK(hipGraphicsUnmapResources(1, &impl_->colorTexResource, 0));
  CUDA_CHECK(hipGraphicsUnmapResources(1, &impl_->depthTexResource, 0));

  // Synchronize
  CUDA_CHECK(hipDeviceSynchronize());
}

}  // namespace gfx
}  // namespace esp

